#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void kuwuhara_filtering(const float4* input, float4* output,
                                   u32 width, u32 height, i32 radius) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < radius || y < radius || x >= width - radius || y >= height - radius)
    return;

  float4 mean[4] = {};
  float4 variance[4] = {};
  int count[4] = {0, 0, 0, 0};

  for (i32 dy = -radius; dy <= radius; dy++) {
    for (i32 dx = -radius; dx <= radius; dx++) {
      int region = (dy <= 0 ? 0 : 2) + (dx > 0 ? 1 : 0);
      float4 col = input[(y + dy) * width + (x + dx)];

      mean[region].x += col.x;
      mean[region].y += col.y;
      mean[region].z += col.z;
      mean[region].w += col.w;

      variance[region].x += col.x * col.x;
      variance[region].y += col.y * col.y;
      variance[region].z += col.z * col.z;
      variance[region].w += col.w * col.w;

      count[region]++;
    }
  }

  float minSigma = INFINITY;
  float4 result = { 0.0, 0.0, 0.0, 1.0 };
  for (i32 i = 0; i < 4; i++) {
    mean[i].x /= count[i];
    mean[i].y /= count[i];
    mean[i].z /= count[i];
    mean[i].w /= count[i];

    variance[i].x = fabsf(variance[i].x / count[i] - mean[i].x * mean[i].x);
    variance[i].y = fabsf(variance[i].y / count[i] - mean[i].y * mean[i].y);
    variance[i].z = fabsf(variance[i].z / count[i] - mean[i].z * mean[i].z);
    variance[i].w = fabsf(variance[i].w / count[i] - mean[i].w * mean[i].w);

    float sigma = variance[i].x + variance[i].y + variance[i].z;
    if (sigma < minSigma) {
      minSigma = sigma;
      result = {mean[i].x, mean[i].y, mean[i].z, 1.0 };
    }
  }

  output[y * width + x] = result;
}